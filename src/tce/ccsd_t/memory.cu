#include "header.h"
#include <map>
#include <set>
using namespace std;

/* #define NO_OPT */

extern "C" {

static int is_init=0;

  static map<int,set<void*> > free_list_gpu, free_list_host;
  static map<void *,int> live_ptrs_gpu, live_ptrs_host;

  static void clearGpuFreeList() {
    for(map<int,set<void*> >::iterator it=free_list_gpu.begin(); 
	it!=free_list_gpu.end(); ++it) {
      for(set<void*>::iterator it2=it->second.begin();
	  it2!=it->second.end(); ++it2) {
	hipFree(*it2);
      }
    }
    free_list_gpu.clear();
  }
  
  static void clearHostFreeList() {
    for(map<int,set<void*> >::iterator it=free_list_host.begin(); 
	it!=free_list_host.end(); ++it) {
      for(set<void*>::iterator it2=it->second.begin();
	  it2!=it->second.end(); ++it2) {
	hipHostFree(*it2);
      }
    }
    free_list_host.clear();
  }

  static int num_resurrections=0, num_morecore=0;

  typedef hipError_t (*mallocfn_t)(void **ptr, size_t bytes);
  static void *morecore(mallocfn_t fn, size_t bytes) {
    void *ptr;
    CUDA_SAFE(fn((void **)&ptr, bytes));
    num_morecore += 1;
    if(ptr==NULL) {
      /*try one more time*/
      clearHostFreeList();
      clearGpuFreeList();
      fn((void **)&ptr, bytes);
    }
    assert(ptr!=NULL); /*We hopefully have a pointer*/
    return ptr;
  }

  static inline void *resurrect_from_free_list(map<int,set<void *> > &free_map,
					size_t bytes, map<void*,int>& liveset) {
    void *ptr;
    num_resurrections +=1 ;
    assert(free_map.find(bytes) != free_map.end());
/*     assert(free_map.find(bytes)->second.size() > 0); */
    set<void *> &st = free_map.find(bytes)->second;
    ptr = *st.begin();
    st.erase(ptr);
    if(st.size()==0)
      free_map.erase(bytes);
    liveset[ptr] = bytes;
    return ptr;
  }

  void initmemmodule_()  {
    is_init=1;
  }

void *getGpuMem(size_t bytes) {
  assert(is_init);
  void *ptr;
#ifdef NO_OPT
  CUDA_SAFE(hipMalloc((void **) &ptr, bytes));
#else
  if(free_list_gpu.find(bytes)!=free_list_gpu.end()) {
    set<void*> &lst = free_list_gpu.find(bytes)->second;
    if(lst.size()!=0) {
      ptr = resurrect_from_free_list(free_list_gpu, bytes, live_ptrs_gpu);
      return ptr;
    }
  }
  else {
    for(map<int,set<void *> >::iterator it=free_list_gpu.begin();
	it != free_list_gpu.end(); ++it) {
      if(it->first >= bytes && it->second.size()>0) {
	ptr = resurrect_from_free_list(free_list_gpu, it->first, live_ptrs_gpu);
	return ptr;
      }
    }
  }
  ptr = morecore(hipMalloc, bytes);
/*   cutilSafeCall(hipMalloc((void **) &ptr, bytes)); */
  live_ptrs_gpu[ptr] = bytes;
#endif
  return ptr;
}

void *getHostMem(size_t bytes) {
  assert(is_init);
  void *ptr;
#ifdef NO_OPT
  CUDA_SAFE(hipHostMalloc((void **) &ptr, bytes));
#else
  if(free_list_host.find(bytes)!=free_list_host.end()) {
    set<void*> &lst = free_list_host.find(bytes)->second;
    if(lst.size()!=0) {
      ptr = resurrect_from_free_list(free_list_host, bytes, live_ptrs_host);
/*       ptr = *lst.begin(); */
/*       lst.erase(lst.begin()); */
/*       live_ptrs_host[ptr] = bytes; */
      return ptr;
    }
  }
  else {
    for(map<int,set<void *> >::iterator it=free_list_host.begin();
	it != free_list_host.end(); ++it) {
      if(it->first >= bytes && it->second.size()>0) {
	ptr = resurrect_from_free_list(free_list_host, it->first, live_ptrs_host);
/* 	set<void*> &lst = it->second; */
/* 	ptr = *lst.begin(); */
/* 	lst.erase(lst.begin()); */
/* 	live_ptrs_gpu[ptr] = bytes; */
	return ptr;
      }
    }
  }
/*   cutilSafeCall(hipHostMalloc((void **) &ptr, bytes)); */
  ptr = morecore(hipHostMalloc, bytes);
  live_ptrs_host[ptr] = bytes;
#endif
  return ptr;
}

void freeHostMem(void *p) {
  int bytes;
  assert(is_init);
#ifdef NO_OPT
  hipHostFree(p);
#else
  assert(live_ptrs_host.find(p) != live_ptrs_host.end());
  bytes = live_ptrs_host[p];
  live_ptrs_host.erase(p);
  free_list_host[bytes].insert(p);
#endif
}

void freeGpuMem(void *p) {
  int bytes;
  assert(is_init);
#ifdef NO_OPT
  hipFree(p);
#else
  assert(live_ptrs_gpu.find(p) != live_ptrs_gpu.end());
  bytes = live_ptrs_gpu[p];
  live_ptrs_gpu.erase(p);
  free_list_gpu[bytes].insert(p);
#endif
}

void finalizememmodule_() {
  assert(is_init);
  is_init = 0;
  
  /*there should be no live pointers*/
  assert(live_ptrs_gpu.size()==0);
  assert(live_ptrs_host.size()==0);

  /*release all freed pointers*/
  clearGpuFreeList();
  clearHostFreeList();
  //printf("num. resurrections=%d \t num. morecore=%d\n", num_resurrections, num_morecore);
}

}

